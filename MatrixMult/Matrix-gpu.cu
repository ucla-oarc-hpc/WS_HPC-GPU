#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

// Kernel for multiplying matrices
__global__ void matrixMulCUDA(int *a, int *b, int *c, int N) {
    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    int tempSum = 0;
    if (ROW < N && COL < N) {
        for (int i = 0; i < N; i++) {
            tempSum += a[ROW * N + i] * b[i * N + COL];
        }
        c[ROW * N + COL] = tempSum;
    }
}

int main() {
    int N = 1024;  // Size of the matrix (1024x1024)
    int SIZE = N * N;

    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;

    size_t bytes = SIZE * sizeof(int);

    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        h_a[i] = rand() % 100;
        h_b[i] = rand() % 100;
    }

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    auto start = std::chrono::high_resolution_clock::now();

    matrixMulCUDA<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();

    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    std::chrono::duration<double, std::milli> elapsed = end - start;
    std::cout << "CUDA Matrix multiplication time: " << elapsed.count() << " ms." << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}